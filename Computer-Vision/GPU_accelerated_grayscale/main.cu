#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <string>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <opencv\cv.h>
#include <opencv\highgui.h>
#include <iostream>

__global__ void rgb_2_grey(unsigned char* greyImage, unsigned char* rgbImage, int rows, int cols)
{
    int rgb_x = blockIdx.x * blockDim.x + threadIdx.x; //x coordinate of pixel
    int rgb_y = blockIdx.y * blockDim.y + threadIdx.y; //y coordinate of pixel

    if ((rgb_x >= cols) && (rgb_y >= rows)) {
        return;
    }

	int gray_adr = rgb_y*cols + rgb_x; //absolute pixel position
	int clr_adr = 3 * gray_adr;       // calculating address for reading RGB values

	if (gray_adr < (rows * cols))
	{
		double gray_val = 0.144*rgbImage[clr_adr] + 0.587*rgbImage[clr_adr + 1] + 0.299*rgbImage[clr_adr + 2];
		greyImage[gray_adr] = (unsigned char)gray_val;
	}
}

//   Kernel Calling Function

extern "C" void gray_parallel(unsigned char* h_in, unsigned char* h_out, int elems, int rows, int cols){

	unsigned char* d_in;
	unsigned char* d_out;
	const int BS = 16;
	const dim3 blockSize(BS, BS);
	const dim3 gridSize((cols / BS) + 1, (rows / BS) + 1);
	hipMalloc((void**)&d_in, elems);
	hipMalloc((void**)&d_out, rows*cols);

	hipMemcpy(d_in, h_in, elems*sizeof(unsigned char), hipMemcpyHostToDevice);
	rgb_2_grey << < gridSize, blockSize >> >(d_out, d_in, rows, cols);

	hipMemcpy(h_out, d_out, rows*cols*sizeof(unsigned char), hipMemcpyDeviceToHost);
	hipFree(d_in);
	hipFree(d_out);
}